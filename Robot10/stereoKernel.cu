#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void stereoKernel(unsigned char* left, unsigned char* right, unsigned char* depth,
                             double maxDistance, int rows, int cols){

// compute the row and col of the pixel to be processed
int col = blockIdx.x*blockDim.x + threadIdx.x;
int row = blockIdx.y*blockDim.y + threadIdx.y;

// put your stereo matching code here
// This code should only be for one pixel
// See the video I posted on acceleration stereo on the GPU

int L  = 9;
int w = (L - 1) / 2;
int ssd = 0;
int max_ssd = INT_MAX;
int diff = 0;
int disparity = 0;
int max_disp = 61;
int baseline = 60;
int focal_length = 700;
int max_distance = 5000;
int distance = 0;

for(int k = 0; k < max_disp; k++){
ssd = 0;
 for (int i = -w; i <= w; i++){
     for (int j = -w; j <= w; j++) {
     diff = left[(row+i)*cols+(col+j)] - right[(row+i)*cols+(col+j-k)];
     ssd += diff*diff;
     } 
 }
if (ssd < max_ssd){
  max_ssd = ssd;
  disparity = k;
}
}

//depth[row*cols+col] = (unsigned int)((disparity * 255) / max_disp);

if (disparity > 0) {
distance = (baseline * focal_length) / disparity;
  if(distance < max_distance)
    depth[row*cols+col] = (unsigned int)((distance * 255 ) / max_distance);
  else 
    depth[row*cols+col] = 255; 
}

else
  depth[row*cols+col] = 255;
}
